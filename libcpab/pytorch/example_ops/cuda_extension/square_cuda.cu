#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Kernel declaration
namespace {

__global__ void square_kernel_forward(float* __restrict__ output, 
                                      const float* __restrict__ input, 
                                      size_t N){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < N){
        output[i] = input[i] * input[i];
    }
    return;
}

__global__ void square_kernel_backward(float* __restrict__ output, 
                                       const float* __restrict__ input, 
                                       size_t N){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < N){
        output[i] = 2.0 * input[i];
    }
    return;
}

} // end namespace

// Kernel launcher declaration
at::Tensor square_cuda_forward(at::Tensor input){
    const auto N = input.numel();
    auto output = at::zeros_like(input);    
    const int blockSize = 512;
    const int numBlocks = (N + blockSize - 1) / blockSize;
    square_kernel_forward<<<numBlocks, blockSize>>>(output.data<float>(), 
                                                    input.data<float>(), 
                                                    N);
    return output;
}

at::Tensor square_cuda_backward(at::Tensor input){
    const auto N = input.numel();
    auto output = at::zeros_like(input);
    const int blockSize = 512;
    const int numBlocks = (N + blockSize - 1) / blockSize;
    square_kernel_backward<<<numBlocks, blockSize>>>(output.data<float>(),
                                                     input.data<float>(),
                                                     N);
    return output;
}